#include "hip/hip_runtime.h"
__global__ void     // calculate A = f(Z + b) in neural network, Z = W * A here
CalcHidden (int nRow, int nCol, double *Z, int ldZ, const double *b, int incb)
{    
    int iRow = threadIdx.x;
    int pos = blockIdx.x * ldZ + threadIdx.x;
    
    if (iRow < nRow) {
        double tmp = Z[pos] + b[iRow * incb];

        Z[pos] = tmp / sqrt(tmp * tmp + 1);
    }

    return;
}
//            -------- blockIdx --------
//            |                         |
//            | threadIdx               |
//            |                         |
//            --------------------------
// =================================================================================

__global__ void     // calculate A = f(Z + b) in neural network, Z = W * A here
CalcOutput (int nCol, double *dZ, int incZ, const double *b)
{    
    int iCol = blockIdx.x * blockDim.x + threadIdx.x;

    if (iCol < nCol) {
        int pos = iCol * incZ;

        Z[pos] += b;
    }

    return;
}

__global__ void
Calc_dZ (int nRow, int nCol, double *Z, int ldZ, const double *dZ)
{

}